
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define BLOCK_SIZE 1024
#define NUM_OF_BANKS 32
#define LOG_NUM_OF_BANKS 5
#define SHIFT_BANK(n) \
    (n + (n >> LOG_NUM_OF_BANKS))


__global__ void prefix_sum(float *in, float *out, float* aux, int noc, int res) {
    __shared__ float temp[2*BLOCK_SIZE];
    int n = BLOCK_SIZE*2;
    if (blockIdx.x == noc - 1) n = res;

    int thid = threadIdx.x;
    // printf("thid: %d, bid: %d, noc: %d\n", thid, blockIdx.x, noc);
    temp[SHIFT_BANK(2*thid)] = in[2*(thid + blockIdx.x*BLOCK_SIZE)];
    temp[SHIFT_BANK(2*thid+1)] = in[2*(thid + blockIdx.x*BLOCK_SIZE)+1];
    
    int offset = 1;
    for (int d = (n >> 1); d > 0; d >>= 1) {
        __syncthreads();
        if (thid < d) {
            int ai = SHIFT_BANK(offset*(2*thid+1) - 1);
            int bi = SHIFT_BANK(offset*(2*thid+2) - 1);
            temp[bi] += temp[ai];
        }
        offset <<= 1;
    }
    
    __syncthreads();
    
    
    if (thid == 0) {
        aux[blockIdx.x] = temp[SHIFT_BANK(n - 1)];
        temp[SHIFT_BANK(n-1)] = 0;
    }

    for (int d = 1; d <= (n >> 1); d <<= 1) {
        offset >>= 1;
        __syncthreads();
        if (thid < d) {
            int ai = SHIFT_BANK(offset*(2*thid+1) - 1);
            int bi = SHIFT_BANK(offset*(2*thid+2) - 1);
            float t = temp[ai] + temp[bi];
            temp[ai] = temp[bi];
            temp[bi] = t;
        }
    }

    
    
    __syncthreads();
    out[2*(threadIdx.x + blockIdx.x * BLOCK_SIZE)] = temp[SHIFT_BANK(2*thid)];
    out[2*(threadIdx.x + blockIdx.x * BLOCK_SIZE) + 1] = temp[SHIFT_BANK(2*thid+1)];
}

__global__ void block_add(float *in, float *out){
    out[2*(threadIdx.x + blockIdx.x * BLOCK_SIZE)] += in[blockIdx.x];
    out[2*(threadIdx.x + blockIdx.x * BLOCK_SIZE) + 1] += in[blockIdx.x];
}

void prefix_sum_cpu_rec(float *in, float *out, int noc, int n) {
    float *aux;
    hipMalloc((void **)&aux, noc*sizeof(float));
    int res = 2*BLOCK_SIZE;
    if (n%(2*BLOCK_SIZE) != 0 || n == 0) res = n%(2*BLOCK_SIZE); 
    prefix_sum<<<noc, BLOCK_SIZE>>>(in, out, aux, noc, res);
    
    hipDeviceSynchronize();
    if (noc == 1) {
        hipFree(aux);
        return;
    }
    float *auxout;
    hipMalloc((void **)&auxout, ((noc + 2*BLOCK_SIZE - 1)/(2*BLOCK_SIZE))*sizeof(float));
    prefix_sum_cpu_rec(aux, auxout, (noc + 2*BLOCK_SIZE - 1)/(2*BLOCK_SIZE), noc);
    block_add<<<noc, BLOCK_SIZE>>>(auxout, out);
    hipDeviceSynchronize();
}

int main() {
    int n = 1 << 20;
    float *in, *out;

    int noc = (n + 2*BLOCK_SIZE - 1)/(2*BLOCK_SIZE);
    hipMalloc((void **)&in, n*sizeof(float));
    hipMalloc((void **)&out, n*sizeof(float));
    
    srand(0);    

    float test[n];
    for (int i = 0; i < n; ++i) {
        float t = (rand()%10)/10.0f;
        test[i] = t;  
    }

    hipMemcpy(in, test, n*sizeof(float), hipMemcpyHostToDevice);
    
    auto start = std::chrono::steady_clock::now();

    prefix_sum_cpu_rec(in,out, noc, n);    

    auto end = std::chrono::steady_clock::now();
    auto diff = end - start;
    std::cout << std::chrono::duration <double, std::milli> (diff).count() << " ms" << std::endl;
    printf("%s\n", hipGetErrorString(hipGetLastError()));
    for (int i = n - 1; i < n; ++i) {
        float a;
        hipMemcpy(&a, out+i, sizeof(float), hipMemcpyDeviceToHost);
        std::cout << a << std::endl;
    }
    
    hipFree(in);
    hipFree(out);
}